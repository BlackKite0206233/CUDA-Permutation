#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <ctime>
#include <cmath>
#include <string>
#include<iostream>

#include "InputCSV.h"
#include "OutputCSV.h"


using namespace std;

__device__ bool Compare(int* set, int* winningSet, int size) {
    int Any = 0;
    for (int i = 0; i < size; i++) {
        if (winningSet[i] > 0) {
            // ordinary compare
            if (set[i] != winningSet[i]) {
                return false;
            }
        } else if (winningSet[i] == -1) {
            // any
            if (Any == 0) {
                Any = set[i];
            } else {
                if (set[i] != Any) {
                    return false;
                }
            }
        }
    }

    return true;
}

// �]�w�C��kernel���üƺؤl
__global__ void SetupCurand(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}
// �]����
__global__ void Simulate(hiprandState *states, int colunmSize, int rowSize, int* reelSets, int reelSetSize, int* winningSets, int winningSetSize, size_t runTimes, size_t* winningSetCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = states[idx];
    int* set;
    set = (int*) malloc(colunmSize*rowSize * sizeof(int));
    for (int t = 0; t < runTimes; t++) {
        for (int col = 0; col < colunmSize; col++) {
            unsigned int rand = hiprand(&localState) % reelSetSize;
            for (int row = 0; row < rowSize; row++) {
                set[row + col*rowSize] = reelSets[(rand + row) % reelSetSize];
            }
        }
        for (int n = 0; n < winningSetSize; n++) {
            if (Compare(set, (winningSets + colunmSize*rowSize*n), colunmSize*rowSize)) {
                atomicAdd(winningSetCount + n, 1);
            }
        }
        // Validate run time.
        //atomicAdd(realRunTimes, 1);
        states[idx] = localState;
    }
    free(set);
};

int main(int argc, char** argv) {
    // �[�J�Ѽ�
    if(argc != 3){ printf(".exe [input file] [output file]\n"); return 1; }
    string intputPath = argv[1];
    string outputPath = argv[2];

    unsigned long cStart = clock();
    InputCSV inputFile(intputPath);
    OutputCSV outputFile(outputPath);
    const unsigned int RUN_TIMES = 100000000;
    const int COLUMN_SIZE = inputFile.getPermutationColumnSize();
    const int REEL_ROW_SIZE = inputFile.getReelRowSize();

    const string *ELEMENTS = inputFile.getPermutationElements();
    const int ELEMENTS_SIZE = inputFile.getPermutationElementsCount();

    const int* REEL_SETS = inputFile.getReelSet();
    const int REEL_SET_SIZE = inputFile.getReelSetSize();

    const int* WINNING_SETS = inputFile.getWinningSets();
    // Size of winning sets(Element count).
    const int WINNING_SETS_SIZE = inputFile.getWinningSetsSize();

    // = sizeof(WINNING_SETS)/sizeof(int)
    const int WINNING_SET_REAL_SIZE = WINNING_SETS_SIZE * COLUMN_SIZE * REEL_ROW_SIZE;

    //---------------------Begin of cuda-----------------------------
    size_t *winningSetCount;
    size_t *dev_winningSetCount;

    int* dev_reelSets;
    int* dev_winningSets;


    // �]�w thread & block.
    unsigned int threads = 10;
    unsigned int blocks = 1000;

    unsigned int NumOfThread = blocks * threads, kernelRunTimes = ceil(RUN_TIMES / NumOfThread);
    printf("Total times: %d\nBlock count: %d\nThread count: %d\nKernelRunTimes: %d\n", RUN_TIMES, blocks, threads, kernelRunTimes);

    // �t�mHost memory.
    winningSetCount = (size_t*) malloc(WINNING_SETS_SIZE * sizeof(size_t));


    // �t�mDevice memory.
    hipMalloc((void**) &dev_winningSetCount, WINNING_SETS_SIZE * sizeof(size_t));


    // Declare reel sets.
    hipMalloc((void**) &dev_reelSets, REEL_SET_SIZE * sizeof(int));
    hipMemcpy(dev_reelSets, REEL_SETS, REEL_SET_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Declare winning sets.
    hipMalloc((void**) &dev_winningSets, WINNING_SET_REAL_SIZE * sizeof(int));
    hipMemcpy(dev_winningSets, WINNING_SETS, WINNING_SET_REAL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Setup random seed for each threads.
    hiprandState* devStates;
    hipMalloc(&devStates, NumOfThread * sizeof(hiprandState));
    SetupCurand << <blocks, threads >> > (devStates, time(NULL));

    // Simulate.
    Simulate <<<blocks, threads >> > (devStates, COLUMN_SIZE, REEL_ROW_SIZE, dev_reelSets, REEL_SET_SIZE, dev_winningSets, WINNING_SETS_SIZE, kernelRunTimes, dev_winningSetCount);

    // Copy device memory to host.
    hipMemcpy(winningSetCount, dev_winningSetCount, WINNING_SETS_SIZE * sizeof(size_t), hipMemcpyDeviceToHost);


    //����Memory.
    hipFree(dev_reelSets);
    hipFree(dev_winningSets);
    hipFree(dev_winningSetCount);

    //---------------------End of cuda-----------------------------

    unsigned long cEnd = clock();
    printf("CUDA run %lu ms.\n", cEnd - cStart);

    printf("Output to %s... \n", outputPath.c_str());

    // ��X
    outputFile.WriteTitle(blocks, threads, RUN_TIMES, RUN_TIMES, cEnd - cStart, ELEMENTS_SIZE, COLUMN_SIZE, REEL_ROW_SIZE);

    //output winning rate ot csv file.
    for (int i = 0; i < WINNING_SETS_SIZE; i++) {
        //[TEMP]
        outputFile.WriteWinningRate(inputFile.getWinningSetName(i), winningSetCount[i], ((double) winningSetCount[i] / RUN_TIMES));
    }



    outputFile.Close();

    delete[] winningSetCount;

    printf("Finish.\n");
    system("PAUSE");
    return 0;
}