#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>
#include <ctime>
#include <cmath>
#include <string>
#include<iostream>

#include "InputCSV.h"
#include "OutputCSV.h"


using namespace std;

__device__ bool Compare(int* set, int* winningSet, int size) {
    int Any = 0;
	for(int i = 0; i < size; i++){
		if(set[i] != -1 && winningSet[i] > 0){
            // ordinary compare
            if (set[i] != winningSet[i]) {
                return false;
            }
        } else if (set[i] != -1 && winningSet[i] == -1) {
            // any
            if (Any == 0) {
                Any = set[i];
            } else {
                if (set[i] != Any) {
                    return false;
                }
            }
        }
    }

    return true;
}

// 設定每個kernel的亂數種子
__global__ void SetupCurand(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}
// 跑模擬
__global__ void Simulate(hiprandState *states, const int colunmSize, const int rowSize, int* reelSets, const int reelSetSize, int* winningSets, int winningSetSize, size_t runTimes, size_t* winningSetCount, const size_t NUM_OF_THREAD) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState localState = states[idx];
    int* set;
    set = (int*) malloc(colunmSize*rowSize * sizeof(int));
    for (int t = 0; t < runTimes; t++) {
        for (int col = 0; col < colunmSize; col++) {
            unsigned int rand = hiprand(&localState) % reelSetSize;
            for (int row = 0; row < rowSize; row++) {
                set[row + col*rowSize] = reelSets[(rand + row) % reelSetSize];
            }
        }
        for (int n = 0; n < winningSetSize; n++) {
            if (Compare(set, (winningSets + colunmSize*rowSize*n), colunmSize*rowSize)) {
                winningSetCount[idx + n * NUM_OF_THREAD] += 1;
            }
        }
        // Validate run time.
        //atomicAdd(realRunTimes, 1);
        states[idx] = localState;
    }
    free(set);
};

int main(int argc, char** argv) {
    // 加入參數
    if(argc != 3){ printf(".exe [input file] [output file]\n"); return 1; }
    string intputPath = argv[1];
    string outputPath = argv[2];

    unsigned long cStart = clock();
    InputCSV inputFile(intputPath);
    OutputCSV outputFile(outputPath);
    const unsigned int RUN_TIMES = 50000000;
    const int COLUMN_SIZE = inputFile.getPermutationColumnSize();
    const int REEL_ROW_SIZE = inputFile.getReelRowSize();

    const string *ELEMENTS = inputFile.getPermutationElements();
    const int ELEMENTS_SIZE = inputFile.getPermutationElementsCount();

    const int* REEL_SETS = inputFile.getReelSet();
    const int REEL_SET_SIZE = inputFile.getReelSetSize();

    const int* WINNING_SETS = inputFile.getWinningSets();
    // Size of winning sets(Element count).
    const int WINNING_SETS_SIZE = inputFile.getWinningSetsSize();

    // = sizeof(WINNING_SETS)/sizeof(int)
    const int WINNING_SET_REAL_SIZE = WINNING_SETS_SIZE * COLUMN_SIZE * REEL_ROW_SIZE;

    //---------------------Begin of cuda-----------------------------
    size_t *winningSetCount;
    size_t *host_winningSetCount;
    size_t *dev_winningSetCount;

    int* dev_reelSets;
    int* dev_winningSets;


    // 設定 thread & block.
    unsigned int threads = 10;
    unsigned int blocks = 10000;

    unsigned int NumOfThread = blocks * threads, kernelRunTimes = ceil(RUN_TIMES / NumOfThread);
    printf("Total times: %d\nBlock count: %d\nThread count: %d\nKernelRunTimes: %d\n", RUN_TIMES, blocks, threads, kernelRunTimes);

    // 配置Host memory.
    winningSetCount = (size_t*) malloc(WINNING_SETS_SIZE * sizeof(size_t));
    host_winningSetCount = (size_t*) malloc(NumOfThread * WINNING_SETS_SIZE * sizeof(size_t));


    // 配置Device memory.
    hipMalloc((void**) &dev_winningSetCount, NumOfThread * WINNING_SETS_SIZE * sizeof(size_t));


    // Declare reel sets.
    hipMalloc((void**) &dev_reelSets, REEL_SET_SIZE * sizeof(int));
    hipMemcpy(dev_reelSets, REEL_SETS, REEL_SET_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Declare winning sets.
    hipMalloc((void**) &dev_winningSets, WINNING_SET_REAL_SIZE * sizeof(int));
    hipMemcpy(dev_winningSets, WINNING_SETS, WINNING_SET_REAL_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Setup random seed for each threads.
    hiprandState* devStates;
    hipMalloc(&devStates, NumOfThread * sizeof(hiprandState));
    SetupCurand << <blocks, threads >> > (devStates, time(NULL));

    // Simulate.
    Simulate <<<blocks, threads >> > (devStates, COLUMN_SIZE, REEL_ROW_SIZE, dev_reelSets, REEL_SET_SIZE, dev_winningSets, WINNING_SETS_SIZE, kernelRunTimes, dev_winningSetCount, NumOfThread);

    // Copy device memory to host.
    hipMemcpy(host_winningSetCount, dev_winningSetCount, NumOfThread * WINNING_SETS_SIZE * sizeof(size_t), hipMemcpyDeviceToHost);


    //釋放Memory.
    hipFree(dev_reelSets);
    hipFree(dev_winningSets);
    hipFree(dev_winningSetCount);

    //---------------------End of cuda-----------------------------
    for (size_t i = 0; i < WINNING_SETS_SIZE; i++) {
        winningSetCount[i] = 0;
        for (size_t t = 0; t < NumOfThread; t++) {
            winningSetCount[i] += host_winningSetCount[i * NumOfThread + t];
        }
    }

    unsigned long cEnd = clock();
    printf("CUDA run %lu ms.\n", cEnd - cStart);

    printf("Output to %s... \n", outputPath.c_str());

    // 輸出
    outputFile.WriteTitle(blocks, threads, RUN_TIMES, RUN_TIMES, cEnd - cStart, ELEMENTS_SIZE, COLUMN_SIZE, REEL_ROW_SIZE);

    //output winning rate ot csv file.
    for (int i = 0; i < WINNING_SETS_SIZE; i++) {
        //[TEMP]
        outputFile.WriteWinningRate(inputFile.getWinningSetName(i), winningSetCount[i], ((double) winningSetCount[i] / RUN_TIMES));
    }



    outputFile.Close();

    delete[] winningSetCount;

    printf("Finish.\n");

    return 0;
}
